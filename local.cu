#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define totaldegrees 180
#define binsperdegree 4
#define threadsperblock 512

// data for the real galaxies will be read into these arrays
float *ra_real, *decl_real;
// number of real galaxies
int    NoofReal;

// data for the simulated random galaxies will be read into these arrays
float *ra_sim, *decl_sim;
// number of simulated random galaxies
int    NoofSim;

///////////

int threadsPerBlock = 512;


// Unit Conversion
const float ARCMIN_TO_RAD = (M_PI / 180.0f) / 60.0f;
const float RAD_TO_DEG = 180.0f / M_PI;

const float BIN_SIZE = 0.25f;
const int NUM_BINS = (int)(90.0f / BIN_SIZE);

//Real - Real
int histogramDD[NUM_BINS] = {0};
//Real - Simulated
int histogramDR[NUM_BINS] = {0};
// Simulated  - Simulated
int histogramRR[NUM_BINS] = {0};

float omega[NUM_BINS] = {0.0f};

float calculateAngularSeparation(float raOneRad, float declOneRad, float raTwoRad, float declTwoRad) {
    return acosf(sinf(declOneRad) * sinf(declTwoRad) +
                 cosf(declOneRad) * cosf(declTwoRad) *
                 cosf(raOneRad - raTwoRad)) * RAD_TO_DEG;
}

__device__ float calculateAngularSeparationKernel(float raOneRad, float declOneRad, float raTwoRad, float declTwoRad) {
    return acosf(sinf(declOneRad) * sinf(declTwoRad) +
                 cosf(declOneRad) * cosf(declTwoRad) *
                 cosf(raOneRad - raTwoRad)) * RAD_TO_DEG;
}

__global__ void fillBinsOptimizedKernel(float *ra_real, float *decl_real, int NoofReal, float *ra_sim, float *decl_sim, int NoofSim, int *histogramDD, int *histogramDR, int *histogramRR) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < NoofReal) {
        float raRealRad = ra_real[i] * ARCMIN_TO_RAD;
        float declRealRad = decl_real[i] * ARCMIN_TO_RAD;

        // Real - Real (histogramDD)
        for (int j = 0; j < NoofReal; j++) {
            float raOtherRad = ra_real[j] * ARCMIN_TO_RAD;
            float declOtherRad = decl_real[j] * ARCMIN_TO_RAD;

            float resDeg = calculateAngularSeparationKernel(raRealRad, declRealRad, raOtherRad, declOtherRad);
            if (resDeg >= 0.0f && resDeg < 90.0f) {
                int binIndex = (int)(resDeg / BIN_SIZE);
                atomicAdd(&histogramDD[binIndex], 1);
            }
        }

        // Real - Sim (histogramDR)
        for (int j = 0; j < NoofSim; j++) {
            float raSimRad = ra_sim[j] * ARCMIN_TO_RAD;
            float declSimRad = decl_sim[j] * ARCMIN_TO_RAD;

            float resDeg = calculateAngularSeparationKernel(raRealRad, declRealRad, raSimRad, declSimRad);
            if (resDeg >= 0.0f && resDeg < 90.0f) {
                int binIndex = (int)(resDeg / BIN_SIZE);
                atomicAdd(&histogramDR[binIndex], 1);
            }
        }
    }
    // Avoid that the index gets out of bounds
    if (i < NoofSim) {
        float raSimRad = ra_sim[i] * ARCMIN_TO_RAD;
        float declSimRad = decl_sim[i] * ARCMIN_TO_RAD;

        for (int j = 0; j < NoofSim; j++) {
            float raOtherRad = ra_sim[j] * ARCMIN_TO_RAD;
            float declOtherRad = decl_sim[j] * ARCMIN_TO_RAD;

            float resDeg = calculateAngularSeparationKernel(raSimRad, declSimRad, raOtherRad, declOtherRad);
            if (resDeg >= 0.0f && resDeg < 90.0f) {
                int binIndex = (int)(resDeg / BIN_SIZE);
                atomicAdd(&histogramRR[binIndex], 1);
            }
        }
    }
}


int fillBinsOptimized() {
    if (ra_real == NULL || decl_real == NULL || ra_sim == NULL || decl_sim == NULL) {
        printf("Error: Null pointer detected in fillBinsOptimized.\n");
        return -1;
    }

    for (int i = 0; i < NoofReal; i++) {
        float raRealRad = ra_real[i] * ARCMIN_TO_RAD;
        float declRealRad = decl_real[i] * ARCMIN_TO_RAD;

        // Real - Real (histogramDD)
        for (int j = 0; j < NoofReal; j++) {
            float raOtherRad = ra_real[j] * ARCMIN_TO_RAD;
            float declOtherRad = decl_real[j] * ARCMIN_TO_RAD;

            float resDeg = calculateAngularSeparation(raRealRad, declRealRad, raOtherRad, declOtherRad);
            if (resDeg >= 0.0f && resDeg < 90.0f) {
                int binIndex = (int)(resDeg / BIN_SIZE);
                histogramDD[binIndex]++;
            }
        }

        // Real - Sim (histogramDR)
        for (int j = 0; j < NoofSim; j++) {
            float raSimRad = ra_sim[j] * ARCMIN_TO_RAD;
            float declSimRad = decl_sim[j] * ARCMIN_TO_RAD;

            float resDeg = calculateAngularSeparation(raRealRad, declRealRad, raSimRad, declSimRad);
            if (resDeg >= 0.0f && resDeg < 90.0f) {
                int binIndex = (int)(resDeg / BIN_SIZE);
                histogramDR[binIndex]++;
            }
        }
    }

    // Simulated - Simulated (histogramRR)
    for (int i = 0; i < NoofSim; i++) {
        float raSimRad = ra_sim[i] * ARCMIN_TO_RAD;
        float declSimRad = decl_sim[i] * ARCMIN_TO_RAD;

        for (int j = 0; j < NoofSim; j++) {
            float raOtherRad = ra_sim[j] * ARCMIN_TO_RAD;
            float declOtherRad = decl_sim[j] * ARCMIN_TO_RAD;

            float resDeg = calculateAngularSeparation(raSimRad, declSimRad, raOtherRad, declOtherRad);
            if (resDeg >= 0.0f && resDeg < 90.0f) {
                int binIndex = (int)(resDeg / BIN_SIZE);
                histogramRR[binIndex]++;
            }
        }
    }
    return 0;
}

int calculateOmega() {
    for (int i = 0; i < NUM_BINS; i++) {
        if (histogramRR[i] != 0) {
            omega[i] = (float)(histogramDD[i] - 2*histogramDR[i] + histogramRR[i]) / histogramRR[i];
        } else {
            omega[i] = 0.0f;
        }
    }
    return 0;
}

int printResults(){
    printf("Bin counts (First 10 bins):\n");

    printf("HistogramDD:\n");
    for (int i = 0; i < 10; i++) printf("Bin %d: %d\n", i, histogramDD[i]);
    printf("HistogramDR:\n");
    for (int i = 0; i < 10; i++) printf("Bin %d: %d\n", i, histogramDR[i]);
    printf("HistogramRR:\n");
    for (int i = 0; i < 10; i++) printf("Bin %d: %d\n", i, histogramRR[i]);
    printf("Omega:\n");
    for (int i = 0; i < 10; i++) printf("Bin %d: %f\n", i, omega[i]);

    return 0;
}


int main(int argc, char *argv[])
{
   int    i;
   int    noofblocks;
   int    readdata(char *argv1, char *argv2);
   int    getDevice(int deviceno);
   long int histogramDRsum, histogramDDsum, histogramRRsum;
   double w;
   double start, end, kerneltime;
   struct timeval _ttime;
   struct timezone _tzone;
   hipError_t myError;

   FILE *outfil;

   if ( argc != 4 ) {printf("Usage: a.out real_data random_data output_data\n");return(-1);}

   if ( getDevice(0) != 0 ) return(-1);

   if ( readdata(argv[1], argv[2]) != 0 ) return(-1);

   // allocate memory on the GPU

    // Memory allocation varibles
    int *histogramDDGPU, *histogramDRGPU, *histogramRRGPU;
    float *raRealGPU, *declRealGPU, *raSimGPU, *declSimGPU;

    // Sizes for memory allocation
    size_t arraybytes = NUM_BINS * sizeof(int);
    size_t realbytes = NoofReal * sizeof(float);
    size_t simbytes = NoofSim * sizeof(float);

    // Memory allocation on the GPU
    hipMalloc(&histogramDDGPU, arraybytes);
    hipMalloc(&histogramDRGPU, arraybytes);
    hipMalloc(&histogramRRGPU, arraybytes);

    hipMalloc(&raRealGPU, realbytes);
    hipMalloc(&declRealGPU, realbytes);

    hipMalloc(&raSimGPU, simbytes);
    hipMalloc(&declSimGPU, simbytes);

    // Copy data to the GPU
    hipMemcpy(raRealGPU, ra_real, realbytes, hipMemcpyHostToDevice);
    hipMemcpy(declRealGPU, decl_real, realbytes, hipMemcpyHostToDevice);
    hipMemcpy(raSimGPU, ra_sim, simbytes, hipMemcpyHostToDevice);
    hipMemcpy(declSimGPU, decl_sim, simbytes, hipMemcpyHostToDevice);
    hipMemcpy(histogramDDGPU, histogramDD, arraybytes, hipMemcpyHostToDevice);
    hipMemcpy(histogramDRGPU, histogramDR, arraybytes, hipMemcpyHostToDevice);
    hipMemcpy(histogramRRGPU, histogramRR, arraybytes, hipMemcpyHostToDevice);

    // Run the kernels on the GPU
    int blocksPerGrid = (NoofReal + threadsPerBlock - 1) / threadsPerBlock;
    fillBinsOptimizedKernel<<<blocksPerGrid, threadsPerBlock>>>(raRealGPU, declRealGPU, NoofReal, raSimGPU, declSimGPU, NoofSim, histogramDDGPU, histogramDRGPU, histogramRRGPU);


    // Copy the results back to the CPU
    hipMemcpy(histogramDD, histogramDDGPU, arraybytes, hipMemcpyDeviceToHost);
    hipMemcpy(histogramDR, histogramDRGPU, arraybytes, hipMemcpyDeviceToHost);
    hipMemcpy(histogramRR, histogramRRGPU, arraybytes, hipMemcpyDeviceToHost);

    // Free the memory on the GPU
    hipFree(histogramDDGPU);
    hipFree(histogramDRGPU);
    hipFree(histogramRRGPU);
    hipFree(raRealGPU);
    hipFree(declRealGPU);
    hipFree(raSimGPU);
    hipFree(declSimGPU);

    // calculate omega values on the CPU, can of course be done on the GPU
    calculateOmega();
    // print the results
    printResults();

   kerneltime = 0.0;
   gettimeofday(&_ttime, &_tzone);
   start = (double)_ttime.tv_sec + (double)_ttime.tv_usec/1000000.;

// run your kernel here

   gettimeofday(&_ttime, &_tzone);
   end = (double)_ttime.tv_sec + (double)_ttime.tv_usec/1000000.;
   kerneltime += end-start;
   printf("   Run time = %.lf secs\n",kerneltime);

   ////

//    fillBinsOptimized();

//    calculateOmega();
//
//    printResults();

    free(ra_real);
    free(decl_real);
    free(ra_sim);
    free(decl_sim);


   return(0);
}


int readdata(char *argv1, char *argv2)
{
  int i,linecount;
  char inbuf[180];
  double ra, dec, phi, theta, dpi;
  FILE *infil;

  printf("   Assuming input data is given in arc minutes!\n");
                          // spherical coordinates phi and theta in radians:
                          // phi   = ra/60.0 * dpi/180.0;
                          // theta = (90.0-dec/60.0)*dpi/180.0;

  dpi = acos(-1.0);
  infil = fopen(argv1,"r");
  if ( infil == NULL ) {printf("Cannot open input file %s\n",argv1);return(-1);}

  // read the number of galaxies in the input file
  int announcednumber;
  if ( fscanf(infil,"%d\n",&announcednumber) != 1 ) {printf(" cannot read file %s\n",argv1);return(-1);}
  linecount =0;
  while ( fgets(inbuf,180,infil) != NULL ) ++linecount;
  rewind(infil);

  if ( linecount == announcednumber ) printf("   %s contains %d galaxies\n",argv1, linecount);
  else
      {
      printf("   %s does not contain %d galaxies but %d\n",argv1, announcednumber,linecount);
      return(-1);
      }

  NoofReal = linecount;
  ra_real   = (float *)calloc(NoofReal,sizeof(float));
  decl_real = (float *)calloc(NoofReal,sizeof(float));

  // skip the number of galaxies in the input file
  if ( fgets(inbuf,180,infil) == NULL ) return(-1);
  i = 0;
  while ( fgets(inbuf,80,infil) != NULL )
      {
      if ( sscanf(inbuf,"%lf %lf",&ra,&dec) != 2 )
         {
         printf("   Cannot read line %d in %s\n",i+1,argv1);
         fclose(infil);
         return(-1);
         }
      ra_real[i]   = (float)ra;
      decl_real[i] = (float)dec;
      ++i;
      }

  fclose(infil);

  if ( i != NoofReal )
      {
      printf("   Cannot read %s correctly\n",argv1);
      return(-1);
      }

  infil = fopen(argv2,"r");
  if ( infil == NULL ) {printf("Cannot open input file %s\n",argv2);return(-1);}

  if ( fscanf(infil,"%d\n",&announcednumber) != 1 ) {printf(" cannot read file %s\n",argv2);return(-1);}
  linecount =0;
  while ( fgets(inbuf,80,infil) != NULL ) ++linecount;
  rewind(infil);

  if ( linecount == announcednumber ) printf("   %s contains %d galaxies\n",argv2, linecount);
  else
      {
      printf("   %s does not contain %d galaxies but %d\n",argv2, announcednumber,linecount);
      return(-1);
      }

  NoofSim = linecount;
  ra_sim   = (float *)calloc(NoofSim,sizeof(float));
  decl_sim = (float *)calloc(NoofSim,sizeof(float));

  // skip the number of galaxies in the input file
  if ( fgets(inbuf,180,infil) == NULL ) return(-1);
  i =0;
  while ( fgets(inbuf,80,infil) != NULL )
      {
      if ( sscanf(inbuf,"%lf %lf",&ra,&dec) != 2 )
         {
         printf("   Cannot read line %d in %s\n",i+1,argv2);
         fclose(infil);
         return(-1);
         }
      ra_sim[i]   = (float)ra;
      decl_sim[i] = (float)dec;
      ++i;
      }

  fclose(infil);

  if ( i != NoofSim )
      {
      printf("   Cannot read %s correctly\n",argv2);
      return(-1);
      }

  return(0);
}




int getDevice(int deviceNo)
{

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  printf("   Found %d CUDA devices\n",deviceCount);
  if ( deviceCount < 0 || deviceCount > 128 ) return(-1);
  int device;
  for (device = 0; device < deviceCount; ++device) {
       hipDeviceProp_t deviceProp;
       hipGetDeviceProperties(&deviceProp, device);
       printf("      Device %s                  device %d\n", deviceProp.name,device);
       printf("         compute capability            =        %d.%d\n", deviceProp.major, deviceProp.minor);
       printf("         totalGlobalMemory             =       %.2lf GB\n", deviceProp.totalGlobalMem/1000000000.0);
       printf("         l2CacheSize                   =   %8d B\n", deviceProp.l2CacheSize);
       printf("         regsPerBlock                  =   %8d\n", deviceProp.regsPerBlock);
       printf("         multiProcessorCount           =   %8d\n", deviceProp.multiProcessorCount);
       printf("         maxThreadsPerMultiprocessor   =   %8d\n", deviceProp.maxThreadsPerMultiProcessor);
       printf("         sharedMemPerBlock             =   %8d B\n", (int)deviceProp.sharedMemPerBlock);
       printf("         warpSize                      =   %8d\n", deviceProp.warpSize);
       printf("         clockRate                     =   %8.2lf MHz\n", deviceProp.clockRate/1000.0);
       printf("         maxThreadsPerBlock            =   %8d\n", deviceProp.maxThreadsPerBlock);
       printf("         asyncEngineCount              =   %8d\n", deviceProp.asyncEngineCount);
       printf("         f to lf performance ratio     =   %8d\n", deviceProp.singleToDoublePrecisionPerfRatio);
       printf("         maxGridSize                   =   %d x %d x %d\n",
                          deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
       printf("         maxThreadsDim in thread block =   %d x %d x %d\n",
                          deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
       printf("         concurrentKernels             =   ");
       if(deviceProp.concurrentKernels==1) printf("     yes\n"); else printf("    no\n");
       printf("         deviceOverlap                 =   %8d\n", deviceProp.deviceOverlap);
       if(deviceProp.deviceOverlap == 1)
       printf("            Concurrently copy memory/execute kernel\n");
       }

    hipSetDevice(deviceNo);
    hipGetDevice(&device);
    if ( device != deviceNo ) printf("   Unable to set device %d, using device %d instead",deviceNo, device);
    else printf("   Using CUDA device %d\n\n", device);

return(0);
}

